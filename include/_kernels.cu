
#include <hip/hip_runtime.h>
#include <stdio.h>



void __global__
maxout(float* best__bo, int* which__bo,
        const float* cands__bop, int B, int O, int P)
{
    int b = blockIdx.x * blockDim.x + threadIdx.x; 
    if (b >= B) return;

    for (int o=0; o < O; ++o)
    {
        which__bo[0] = 0;
        best__bo[0] = cands__bop[0];
        cands__bop += 1;
        for (int p=1; p < P; ++p)
	{
            if (cands__bop[0] > best__bo[0])
	    {
                which__bo[0] = p;
                best__bo[0] = cands__bop[0];
	    }
            cands__bop += 1;
	}
        best__bo += 1;
        which__bo += 1;
    }
}



void __global__
mean_pool(float* means__bo,
    const float* X__to, const int* lengths__b, int B, int T, int O)
{
	// Each CUDA block computes means of a batch of concatenated sequences, using the lengths. 
    	int bid = blockIdx.x;
	if(threadIdx.x>=O)
		return;
	__shared__ float local_means[512];	// can be bigger, depends on dimensions

	// At each step it keeps track of the total length of all previous batches (even those processed 
	// other CUDA blocks)
	int prevLengths = 0;
	for(int i = 0; i<bid; i++)
		prevLengths+=lengths__b[i];

	// Batch-items are processed by a fixed number of launched CUDA blocks
	// with a step equal to the total number gridDim.x
    	for(int step = bid; step < B; step += gridDim.x )
	{
		int lengthOfBatch = lengths__b[step];
		int batchStarts = prevLengths*O; 
		float scale = 1.0/(float)lengthOfBatch;
		local_means[threadIdx.x] = 0.0;

		for (int i = batchStarts + threadIdx.x; i < batchStarts+(lengthOfBatch*O) ; i += O)
			local_means[threadIdx.x] += X__to[i]*scale;
		
		__syncthreads();	// Block-wise synchronization

		means__bo[step*O + threadIdx.x] = local_means[threadIdx.x];

		// prepare prevLength for next steps
		for(int i = step; i<step+gridDim.x; i++)
			prevLengths+=lengths__b[i];
	}
}







void __global__
max_pool(float* maxes__bo, int* which__bo,
    const float* X__to, const int* lengths__b, int B, int T, int O)
{
	// Each CUDA block computes maxes of a batch of concatenated sequences, using the lengths. 
    	int bid = blockIdx.x;
	if(threadIdx.x>=O)
		return;
	__shared__ float local_maxes[512];		// take advantage of faster local memory
	__shared__ short local_which[512];

	// At each step block keeps track of the total length of all previous batches (even those processed 
	// other CUDA blocks)
	int prevLengths = 0;
	for(int i = 0; i<bid; i++)
		prevLengths+=lengths__b[i];

	// Batch-items are processed by a fixed number of launched CUDA blocks
	// with a step equal to the total number gridDim.x
    	for(int step = bid; step < B; step += gridDim.x )
	{
		int lengthOfBatch = lengths__b[step];
		int batchStarts = prevLengths*O; 

		local_maxes[threadIdx.x] = X__to[batchStarts+threadIdx.x];
		local_which[threadIdx.x] = 0;
		short j=1;	// the word index in a doc

		for (int i = batchStarts+O+threadIdx.x; i < batchStarts+(lengthOfBatch*O) ; i += O)
		{
			if(X__to[i]>local_maxes[threadIdx.x])
			{
				local_maxes[threadIdx.x] =  X__to[i];
				local_which[threadIdx.x] = j;
			}
			j++; 
		}
		__syncthreads();	// Block-wise synchronization

		maxes__bo[step*O + threadIdx.x] = local_maxes[threadIdx.x];
		which__bo[step*O + threadIdx.x] = local_which[threadIdx.x];

		// prepare prevLength for next steps
		for(int i = step; i<step+gridDim.x; i++)
			prevLengths+=lengths__b[i];
	}
}


void __global__
backprop_mean_pool(float* dX__to, const float* d_means__bo, const int* lengths__b,
    int B, int T, int O)
{
	// Each CUDA block computes maxes of a batch of concatenated sequences, using the lengths. 
    	int bid = blockIdx.x;
	if(threadIdx.x>=O)
		return;

	__shared__ float local_means[512];		// can be bigger, depends on dimensions

	// At each step it keeps track of the total length of all previous batches (even those processed 
	// other CUDA blocks)
	int prevLengths = 0;
	for(int i = 0; i<bid; i++)
		prevLengths+=lengths__b[i];

    	for(int step = bid; step < B; step += gridDim.x )
	{
		int lengthOfBatch = lengths__b[step];
		int batchStarts = prevLengths*O; 
		float scale = 1.0/(float)lengthOfBatch;
		local_means[threadIdx.x] = d_means__bo[step*O+threadIdx.x]*scale;

		for (int i = batchStarts + threadIdx.x; i < batchStarts+(lengthOfBatch*O) ; i += O)
			dX__to[i] = local_means[threadIdx.x];

		// prepare prevLength for next steps
		for(int i = step; i<step+gridDim.x; i++)
			prevLengths+=lengths__b[i];		
	}
}



void __global__
backprop_max_pool(float* dX__to,
    const float* d_maxes__bo, const int* which__bo, const int* lengths__b, int B, int T, int O)
{
	// Each CUDA block computes maxes of a batch of concatenated sequences, using the lengths. 
    	int bid = blockIdx.x;
	if(threadIdx.x>=O)
		return;
	__shared__ float local_maxes[512];		// can be bigger, depends on dimensions
	__shared__ short local_which[512];
	int prevLengths = 0;
	for(int i = 0; i<bid; i++)
		prevLengths+=lengths__b[i];

    	for(int step = bid; step < B; step += gridDim.x )
	{
		int lengthOfBatch = lengths__b[step];
		int batchStarts = prevLengths*O; 

		local_maxes[threadIdx.x] = d_maxes__bo[step*O+threadIdx.x];
		local_which[threadIdx.x] = which__bo[step*O+threadIdx.x];
		short j=0;	// the word index in a doc

		for (int i = batchStarts+threadIdx.x; i < batchStarts+(lengthOfBatch*O) ; i += O)
		{
			if(local_which[threadIdx.x]==j)
			{
				dX__to[i] =  local_maxes[threadIdx.x];
			}
			else
				dX__to[i]=0;
			j++; 
		}

		// prepare prevLength for next steps
		for(int i = step; i<step+gridDim.x; i++)
			prevLengths+=lengths__b[i];
	}
}


